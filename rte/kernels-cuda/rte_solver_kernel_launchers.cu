#include "rte_kernel_launcher_cuda.h"

namespace
{
    #include "rte_solver_kernels.cu"
}

template<typename TF>
void lw_solver_noscat_gaussquad(
        const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const int nmus,
        const TF* ds, const TF* weights, const TF* tau, const TF* lay_source,
        const TF* lev_source_inc, const TF* lev_source_dec, const TF* sfc_emis,
        const TF* sfc_src, TF* flux_up, TF* flux_dn,
        const TF* sfc_src_jac, TF* flux_up_jac)
{
    TF eps = std::numeric_limits<TF>::epsilon();

    const int sfc_size = ncol * ngpt;
    const int vol_size = sfc_size * nlay;
    const int flx_size = sfc_size * (nlay + 1);

    TF* source_sfc = Tools_gpu::allocate_gpu<TF>(sfc_size);
    TF* source_sfc_jac = Tools_gpu::allocate_gpu<TF>(sfc_size);
    TF* sfc_albedo = Tools_gpu::allocate_gpu<TF>(sfc_size);
    TF* tau_loc = Tools_gpu::allocate_gpu<TF>(vol_size);
    TF* trans = Tools_gpu::allocate_gpu<TF>(vol_size);
    TF* source_dn = Tools_gpu::allocate_gpu<TF>(vol_size);
    TF* source_up = Tools_gpu::allocate_gpu<TF>(vol_size);
    TF* radn_dn = Tools_gpu::allocate_gpu<TF>(flx_size);
    TF* radn_up = Tools_gpu::allocate_gpu<TF>(flx_size);
    TF* radn_up_jac = Tools_gpu::allocate_gpu<TF>(flx_size);


    // Running some permutations of block sizes.
    /*`
    {
        std::cout << "TUNING lw_solver_noscat_gaussquad_kernel" << std::endl;
        std::vector<std::pair<int, int>> col_gpt_combis;
        std::vector<int> cols{ 1, 1, 2, 4, 8, 16, 32, 64, 128, 256, 512};
        std::vector<int> gpts{ 1, 2, 4, 8, 16, 32, 64, 128};
        for (const int igpt : gpts)
            for (const int icol : cols)
                col_gpt_combis.emplace_back(icol, igpt);

        // Create tmp arrays to write output to.
        Array_gpu<TF,3> flux_up_tmp{flux_up}, flux_dn_tmp{flux_dn}, flux_up_jac_tmp{flux_up_jac};

        for (const auto& p : col_gpt_combis)
        {
            std::cout << "(" << p.first << ", " << p.second << "): ";

            const int block_col2d = p.first;
            const int block_gpt2d = p.second;

            const int grid_col2d = ncol/block_col2d + (ncol%block_col2d > 0);
            const int grid_gpt2d = ngpt/block_gpt2d + (ngpt%block_gpt2d > 0);

            dim3 grid_gpu2d(grid_col2d, grid_gpt2d);
            dim3 block_gpu2d(block_col2d, block_gpt2d);

            // Warm it up.
            lw_solver_noscat_gaussquad_kernel<<<grid_gpu2d, block_gpu2d>>>(
                    ncol, nlay, ngpt, eps, top_at_1, nmus, ds, weights, tau, lay_source,
                    lev_source_inc, lev_source_dec, sfc_emis, sfc_src, radn_up,
                    radn_dn, sfc_src_jac, radn_up_jac, tau_loc, trans, source_dn, source_up,
                    source_sfc, sfc_albedo, source_sfc_jac, flux_up_tmp, flux_dn_tmp, flux_up_jac_tmp);

            hipEvent_t start;
            hipEvent_t stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            const int n_samples = 10;

            hipEventRecord(start, 0);
            for (int i=0; i<n_samples; ++i)
                lw_solver_noscat_gaussquad_kernel<<<grid_gpu2d, block_gpu2d>>>(
                        ncol, nlay, ngpt, eps, top_at_1, nmus, ds, weights, tau, lay_source,
                        lev_source_inc, lev_source_dec, sfc_emis, sfc_src, radn_up,
                        radn_dn, sfc_src_jac, radn_up_jac, tau_loc, trans, source_dn, source_up,
                        source_sfc, sfc_albedo, source_sfc_jac, flux_up_tmp, flux_dn_tmp, flux_up_jac_tmp);
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float duration = 0.f;
            hipEventElapsedTime(&duration, start, stop);

            std::cout << std::setprecision(10) << duration << " (ns), check: " << flux_up_tmp({ncol, nlay+1, ngpt}) << ", ";

            // Check whether kernel has succeeded;
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
                std::cout << hipGetErrorString(err) << std::endl;
            else
                std::cout << std::endl;
        }

        std::cout << "STOP TUNING lw_solver_noscat_gaussquad_kernel" << std::endl;
    }
    */
    // End of performance tuning.

    const int block_col2d = 64;
    const int block_gpt2d = 2;

    const int grid_col2d = ncol/block_col2d + (ncol%block_col2d > 0);
    const int grid_gpt2d = ngpt/block_gpt2d + (ngpt%block_gpt2d > 0);

    dim3 grid_gpu2d(grid_col2d, grid_gpt2d);
    dim3 block_gpu2d(block_col2d, block_gpt2d);

    const int block_col3d = 96;
    const int block_lay3d = 1;
    const int block_gpt3d = 1;

    const int grid_col3d = ncol/block_col3d + (ncol%block_col3d > 0);
    const int grid_lay3d = (nlay+1)/block_lay3d + ((nlay+1)%block_lay3d > 0);
    const int grid_gpt3d = ngpt/block_gpt3d + (ngpt%block_gpt3d > 0);

    dim3 grid_gpu3d(grid_col3d, grid_lay3d, grid_gpt3d);
    dim3 block_gpu3d(block_col3d, block_lay3d, block_gpt3d);

    const int top_level = top_at_1 ? 0 : nlay;

    lw_solver_noscat_step1_kernel<<<grid_gpu3d, block_gpu3d>>>(
            ncol, nlay, ngpt, eps, top_at_1, ds, weights, tau, lay_source,
            lev_source_inc, lev_source_dec, sfc_emis, sfc_src, flux_up, flux_dn, sfc_src_jac,
            flux_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);

    lw_solver_noscat_step2_kernel<<<grid_gpu2d, block_gpu2d>>>(
            ncol, nlay, ngpt, eps, top_at_1, ds, weights, tau, lay_source,
            lev_source_inc, lev_source_dec, sfc_emis, sfc_src, flux_up, flux_dn, sfc_src_jac,
            flux_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);
        
    lw_solver_noscat_step3_kernel<<<grid_gpu3d, block_gpu3d>>>(
            ncol, nlay, ngpt, eps, top_at_1, ds, weights, tau, lay_source,
            lev_source_inc, lev_source_dec, sfc_emis, sfc_src, flux_up, flux_dn, sfc_src_jac,
            flux_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);
        
    apply_BC_kernel_lw<<<grid_gpu2d, block_gpu2d>>>(top_level, ncol, nlay, ngpt, top_at_1, flux_dn, radn_dn);

    if (nmus > 1)
    {
        for (int imu=1; imu<nmus; ++imu)
        {
            lw_solver_noscat_step1_kernel<<<grid_gpu3d, block_gpu3d>>>(
                    ncol, nlay, ngpt, eps, top_at_1, ds+imu, weights+imu, tau, lay_source,
                    lev_source_inc, lev_source_dec, sfc_emis, sfc_src, radn_up, radn_dn, sfc_src_jac,
                    radn_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);

            lw_solver_noscat_step2_kernel<<<grid_gpu2d, block_gpu2d>>>(
                    ncol, nlay, ngpt, eps, top_at_1, ds+imu, weights+imu, tau, lay_source,
                    lev_source_inc, lev_source_dec, sfc_emis, sfc_src, radn_up, radn_dn, sfc_src_jac,
                    radn_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);

            lw_solver_noscat_step3_kernel<<<grid_gpu3d, block_gpu3d>>>(
                    ncol, nlay, ngpt, eps, top_at_1, ds+imu, weights+imu, tau, lay_source,
                    lev_source_inc, lev_source_dec, sfc_emis, sfc_src, radn_up, radn_dn, sfc_src_jac,
                    radn_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);

            add_fluxes_kernel<<<grid_gpu3d, block_gpu3d>>>(
                    ncol, nlay+1, ngpt,
                    radn_up, radn_dn, radn_up_jac,
                    flux_up, flux_dn, flux_up_jac);
        }
    }

    Tools_gpu::free_gpu<TF>(source_sfc);
    Tools_gpu::free_gpu<TF>(source_sfc_jac);
    Tools_gpu::free_gpu<TF>(sfc_albedo);
    Tools_gpu::free_gpu<TF>(tau_loc);
    Tools_gpu::free_gpu<TF>(trans);
    Tools_gpu::free_gpu<TF>(source_dn);
    Tools_gpu::free_gpu<TF>(source_up);
    Tools_gpu::free_gpu<TF>(radn_dn);
    Tools_gpu::free_gpu<TF>(radn_up);
    Tools_gpu::free_gpu<TF>(radn_up_jac);
}


extern "C" 
{
    void lw_solver_noscat_gaussquad_wrapper_(const int* ncol, const int* nlay, const int* ngpt, const BOOL_TYPE* top_at_1, const int* nmus,
    const double* ds, const double* weights, const double* tau, const double* lay_source,
    const double* lev_source_inc, const double* lev_source_dec, const double* sfc_emis,
    const double* sfc_src, double* flux_up, double* flux_dn,
    const double* sfc_src_jac, double* flux_up_jac)
    {
        lw_solver_noscat_gaussquad(*ncol, *nlay, *ngpt, *top_at_1, *nmus,
                                ds, weights, tau, lay_source, 
                                lev_source_inc, lev_source_dec, sfc_emis, sfc_src,
                                flux_up, flux_dn, sfc_src_jac, flux_up_jac);
    }
}
